#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <stdint.h>
#include <iterator>
#include<algorithm>
#include <float.h>
#include <math.h>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <iostream>
#include <string>
#include <bitset>
#include <time.h>
#include <map>
#include <vector>
#include <set>
#include<list>
#include<random>


int main(int argc, char* argv[])
{
    int matrixSize = 1024; // square matrix matrixSize * matrixSize
    int numElements = matrixSize * matrixSize;

    printf("done\n" );


    return 0;
}
/***
// Allocate host memory
float *h_A = (float *)malloc(numElements * sizeof(float));
float *h_B = (float *)malloc(numElements * sizeof(float));
float *h_C = (float *)malloc(numElements * sizeof(float));

// Initialize the host input matrixs
for (int i = 0; i < numElements; ++i)
{
    h_A[i] = rand()/(float)RAND_MAX;
    h_B[i] = rand()/(float)RAND_MAX;
}

// Allocate the device input matrix A
float *d_A, *d_B, *d_C;

hipMalloc(&d_A, numElements * sizeof(float));
hipMalloc(&d_B, numElements * sizeof(float));
hipMalloc(&d_C, numElements * sizeof(float));

// Copy the host input matrixs A and B in host memory to the device input matrixs in
hipMemcpy(d_A, h_A, numElements * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, numElements * sizeof(float), hipMemcpyHostToDevice);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds = 0;

// Option 1: 2D grid of 2D thread blocks 16x16 (OK)
{
int threadsPerBlockDim = 16;
int gridDimSize = (matrixSize + threadsPerBlockDim - 1) / threadsPerBlockDim;

dim3 blockSize(threadsPerBlockDim, threadsPerBlockDim);
dim3 gridSize (gridDimSize, gridDimSize);

printf("CUDA kernel launch with %dx%d blocks of %dx%d threads\n", gridDimSize, gridDimSize, threadsPerBlockDim, threadsPerBlockDim);

hipEventRecord(start);

matrixAddv1<<<gridSize, blockSize>>>(d_A, d_B, d_C, matrixSize);

hipEventRecord(stop);
hipEventSynchronize(stop);

hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU option 1 time to sum the matrixes %f ms\n", milliseconds);

// Copy the device result matrix in device memory to the host result matrix
hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

hipError_t hipError_t = hipGetLastError();

if(hipError_t != hipSuccess)
{
    fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
    exit(EXIT_FAILURE);
}

// Verify that the result matrix is correct
for (int i = 0; i < numElements; i++)
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    {
        fprintf(stderr, "Result verification failed at element %d!\n", i);
        exit(EXIT_FAILURE);
    }

printf("Sum of the matrixes was OK\n");
}


// Option 2: 1D grid of 1D thread blocks 1x256 (INEFFICIENT ON PURPOSE), multiple memory transactions!!
{
int threadsPerBlock = 256;
int gridDim = (numElements + threadsPerBlock - 1) / threadsPerBlock; // the dimensionality per grid dimension cannot be larger than 65536 for GPUs using CC 2.0

dim3 blocksize(1, threadsPerBlock);

printf("CUDA kernel launch with %d blocks of 1x%d threads\n", gridDim, threadsPerBlock);

hipEventRecord(start);

matrixAddv2v3<<<gridDim, blocksize>>>(d_A, d_B, d_C, numElements);

hipEventRecord(stop);
hipEventSynchronize(stop);

hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU option 2 time to sum the matrixes %f ms\n", milliseconds);

// Copy the device result matrix in device memory to the host result matrix
hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

hipError_t hipError_t = hipGetLastError();

if(hipError_t != hipSuccess)
{
    fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
    exit(EXIT_FAILURE);
}

// Verify that the result matrix is correct
for (int i = 0; i < numElements; i++)
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    {
        fprintf(stderr, "Result verification failed at element %d!\n", i);
        exit(EXIT_FAILURE);
    }

printf("Sum of the matrixes was OK\n");
}

// Option 3: 1D grid of 1D thread blocks (MOST EFFICIENT), smaller number of larger transactions
{
int threadsPerBlock = 256;
int gridDim = (numElements + threadsPerBlock - 1) / threadsPerBlock; // the dimensionality per grid dimension cannot be larger than 65536 for GPUs using CC 2.0

printf("CUDA kernel launch with %d blocks of %dx1 threads\n", gridDim, threadsPerBlock);

hipEventRecord(start);

matrixAddv2v3<<<gridDim, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

hipEventRecord(stop);
hipEventSynchronize(stop);

hipEventElapsedTime(&milliseconds, start, stop);
printf("GPU option 3 time to sum the matrixes %f ms\n", milliseconds);

// Copy the device result matrix in device memory to the host result matrix
hipMemcpy(h_C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

hipError_t hipError_t = hipGetLastError();

if(hipError_t != hipSuccess)
{
    fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
    exit(EXIT_FAILURE);
}

// Verify that the result matrix is correct
for (int i = 0; i < numElements; i++)
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
    {
        fprintf(stderr, "Result verification failed at element %d!\n", i);
        exit(EXIT_FAILURE);
    }

printf("Sum of the matrixes was OK\n");
}

// Compute CPU time


hipEventRecord(start);

for (int i = 0; i < numElements; i++)
h_C[i] = h_A[i] + h_B[i];

hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("CPU time to sum the matrixes %f ms\n", milliseconds);

// Free device global memory
hipFree(d_A);
hipFree(d_B);
hipFree(d_C);

// Free host memory
free(h_A);
free(h_B);
free(h_C);


***/
